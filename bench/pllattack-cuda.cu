/**
 * mlp: memory-level-parallelism (MLP) detector
 *
 * Copyright (C) 2015  Heechul Yun <heechul.yun@ku.edu> 
 *
 * This file is distributed under the University of Illinois Open Source
 * License. See LICENSE.TXT for details.
 *
 * Usage:
 *   $ sudo ./attacker4-coffeelake -m 65536 -l 12 
 */ 

/**************************************************************************
 * Conditional Compilation Options
 **************************************************************************/

/**************************************************************************
 * Included Files
 **************************************************************************/

#include <hip/hip_runtime.h>
#include <iostream>     // std::cout
#include <algorithm>    // std::random_shuffle
#include <vector>       // std::vector
#include <ctime>        // std::time
#include <cstdlib>      // std::rand, std::srand

#include <sched.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <unistd.h>
#include <sys/time.h>
#include <inttypes.h>
#include <sys/mman.h>
#include <sys/types.h>
#include <fcntl.h>
#include <errno.h>
#include <sys/time.h>
#include <sys/resource.h>
#include <assert.h>


/**************************************************************************
 * Public Definitions
 **************************************************************************/
#define MAX_MLP 256
#define CACHE_LINE_SIZE 64
#define DEFAULT_ALLOC_SIZE_KB 65536
#define DEFAULT_ITER 100

#define PAGE_SHIFT 12

#define MAX_COLORS 64

/**************************************************************************
 * Public Types
 **************************************************************************/

/**************************************************************************
 * Global Variables
 **************************************************************************/
static int g_mem_size = (DEFAULT_ALLOC_SIZE_KB*1024);
static int* next;

static int g_debug = 0;
static int g_color[MAX_COLORS]; // not assigned
static int g_color_cnt = 0;

/**************************************************************************
 * Public Function Prototypes
 **************************************************************************/
uint64_t get_elapsed(struct timespec *start, struct timespec *end)
{
	uint64_t dur;

	dur = ((uint64_t)end->tv_sec * 1000000000 + end->tv_nsec) - 
		((uint64_t)start->tv_sec * 1000000000 + start->tv_nsec);
	return dur;
}

// ----------------------------------------------
long utime()
{
	struct timeval tv;
	gettimeofday(&tv, NULL);

	return (tv.tv_sec) * 1000 + (tv.tv_usec) / 1000;
}

uint64_t nstime()
{
	struct timespec ts;
	clock_gettime(CLOCK_REALTIME, &ts);
	return ts.tv_sec * 1000000000 + ts.tv_nsec;
}


#define bit(addr,x) 	((addr >> (x)) & 0x1)
int paddr_to_color(unsigned long addr)
{
	return ((bit(addr, 6)^bit(addr,13))<<3|
		(bit(addr,14)^bit(addr,17))<<2|
		(bit(addr,15)^bit(addr,18))<<1|
		(bit(addr,16)^bit(addr,19)));
}

// ---------------------------------------------------------------------------
size_t libkdump_virt_to_phys(size_t virtual_address) {
  static int pagemap = -1;
  if (pagemap == -1) {
    pagemap = open("/proc/self/pagemap", O_RDONLY);
    if (pagemap < 0) {
      errno = EPERM;
      return 0;
    }
  }
  uint64_t value;
  int got = pread(pagemap, &value, 8, (virtual_address / 0x1000) * 8);
  if (got != 8) {
    errno = EPERM;
    return 0;
  }
  uint64_t page_frame_number = value & ((1ULL << 54) - 1);
  if (page_frame_number == 0) {
    errno = EPERM;
    return 0;
  }
  return page_frame_number * 0x1000 + virtual_address % 0x1000;
}


/**************************************************************************
 * Implementation
 **************************************************************************/


__global__
void gpuwrite(int* array, int *next, long iter)
{
    const uint64_t tid = blockIdx.x*blockDim.x + threadIdx.x;
    printf("next[%d]=%d iter=%d\n", (int)tid, (int)next[tid], (int)iter);
    for (long r = 0; r < iter; r++) {
        array[next[tid]+1] = 0xff;
	// if (r % 1000 == 0) printf("r=%d\n", r);
        next[tid] = array[next[tid]];
    }
}

int main(int argc, char* argv[])
{
	struct sched_param param;
        cpu_set_t cmask;
	int num_processors;
	int cpuid = 0;

	int *memchunk = NULL;
	int opt, prio;
	int i;

	long repeat = DEFAULT_ITER;
	int mlp = 1;
	struct timespec start, end;

	std::srand (0);
	std::vector<int> myvector;

	/*
	 * get command line options 
	 */
	while ((opt = getopt(argc, argv, "m:a:c:d:e:b:i:l:hx")) != -1) {
		switch (opt) {
		case 'm': /* set memory size */
			g_mem_size = 1024 * strtol(optarg, NULL, 0);
			break;
		case 'c': /* set CPU affinity */
			cpuid = strtol(optarg, NULL, 0);
			fprintf(stderr, "cpuid: %d\n", cpuid);
			num_processors = sysconf(_SC_NPROCESSORS_CONF);
			CPU_ZERO(&cmask);
			CPU_SET(cpuid % num_processors, &cmask);
			if (sched_setaffinity(0, num_processors, &cmask) < 0) {
				perror("error");
				exit(1);
			}
			else
				fprintf(stderr, "assigned to cpu %d\n", cpuid);
			break;
		case 'd': /* debug */
			g_debug = strtol(optarg, NULL, 0);
			break;
		case 'e': /* select color (dram bank) */
			g_color[g_color_cnt++] = strtol(optarg, NULL, 0);
			break;	
		case 'p': /* set priority */
			prio = strtol(optarg, NULL, 0);
			if (setpriority(PRIO_PROCESS, 0, prio) < 0)
				perror("error");
			else
				fprintf(stderr, "assigned priority %d\n", prio);
			break;
		case 'i': /* iterations */
			repeat = strtol(optarg, NULL, 0);
			fprintf(stderr, "repeat=%ld\n", repeat);
			break;
		case 'l': /* MLP */
			mlp = strtol(optarg, NULL, 0);
			fprintf(stderr, "MLP=%d\n", mlp);
			break;
		}

	}

	printf("sizeof(unsigned long): %d\n", (int)sizeof(unsigned long));

	printf("\n");
	if (g_color_cnt) {
		printf("Colors: ");
		for (int i = 0; i < g_color_cnt; i++) {
			printf("%d ", g_color[i]);
		}
		printf("\n");
	}
	
	srand(0);

	int ws = 0;
	int orig_ws = (g_mem_size / CACHE_LINE_SIZE);

	printf("orig_ws: %d  mlp: %d\n", orig_ws, mlp);
	
	clock_gettime(CLOCK_REALTIME, &start);

	
	hipMallocManaged(&memchunk, g_mem_size);
	hipMallocManaged(&next, MAX_MLP * sizeof(int));
	
	/* initialize data */
	memset(memchunk, 0, g_mem_size);

	// set some values:
	int page_size = 1<<PAGE_SHIFT;

	ulong vaddr = 0;
	ulong paddr = 0;
	for (int i=0; i<orig_ws; i++) {
		vaddr = (ulong)&memchunk[i*CACHE_LINE_SIZE/4];
		if (i % (page_size/CACHE_LINE_SIZE) == 0) {
			paddr = (ulong)libkdump_virt_to_phys(vaddr);

			if (g_debug)
				printf("vaddr: %p padddr: %p color: %d\n",
				       (void *)vaddr, (void *)paddr,
				       paddr_to_color(paddr));
		} else
			paddr = paddr + CACHE_LINE_SIZE;
		
		if (g_color_cnt > 0) {
			/* use coloring */
			for (int j = 0; j < g_color_cnt; j++) {
				if (paddr_to_color(paddr) == g_color[j]) {
					myvector.push_back(i);
				}
			}

		} else {
			/* not using coloring */
			myvector.push_back(i);			
		}
	}

	// using built-in random generator:
	std::random_shuffle (myvector.begin(), myvector.end() );

	// update the workingset size
	ws = myvector.size() / mlp * mlp; 

	printf("new ws: %d\n", ws);
	int list_len = ws / mlp;
	
	for (i = 0; i < ws; i++) {
		int l = i / list_len;
		int curr_idx = myvector[i] * CACHE_LINE_SIZE / 4;
		int next_idx = myvector[i+1] * CACHE_LINE_SIZE / 4;
		if ((i+1) % list_len == 0)
			next_idx = myvector[i/list_len*list_len] * CACHE_LINE_SIZE / 4;
		memchunk[curr_idx] = next_idx;
		
		if (i % list_len == 0) {
			next[l] = curr_idx;
			// printf("next[%d]  %d\n", l,  next[l]);
		}
		
		// printf("%8d ->%8d\n", myvector[i], next_idx*4/CACHE_LINE_SIZE);
	}
	
        param.sched_priority = 1;
        if(sched_setscheduler(0, SCHED_FIFO, &param) == -1) {
		perror("sched_setscheduler failed");
        }

	clock_gettime(CLOCK_REALTIME, &end);
	printf("Init took %.0f us\n", (double) get_elapsed(&start, &end)/1000);

	long naccess;
	int64_t nsdiff;
	double  avglat;
	int total_ws;

	int blockSize = 256;
	int numBlocks = (mlp + blockSize - 1) / blockSize;
	// long iter = repeat * list_len; 
	printf("launch the gpu kernel\n");
	clock_gettime(CLOCK_REALTIME, &start);
	gpuwrite<<<numBlocks, blockSize>>>(memchunk, next, repeat);
	hipDeviceSynchronize();
	clock_gettime(CLOCK_REALTIME, &end);
	fprintf(stderr, "gpu kernel finishes\n");

	naccess = mlp * repeat;
	nsdiff = get_elapsed(&start, &end);
	avglat = (double)nsdiff/naccess;

	printf("alloc. size: %d (%d KB)\n", g_mem_size, g_mem_size/1024);
	total_ws =  ws * CACHE_LINE_SIZE;
	printf("ws size: %d (%d KB)\n", total_ws, total_ws / 1024);
	printf("list_len: %d (%d KB)\n", list_len, list_len * CACHE_LINE_SIZE / 1024);
	printf("mlp: %d\n", mlp);
	printf("duration %.0f ns (%.2f sec), #access %ld\n",
	       (double)nsdiff, (double)nsdiff/1000000000, naccess);
	printf("Avg. latency %.2f ns\n", avglat);	
	printf("bandwidth %.2f MB/s\n", (double)64*1000*naccess/nsdiff);
	
	return 0;
}
